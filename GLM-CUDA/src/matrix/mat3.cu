#include "../../include/matrix.cuh"

#include "../../include/utility.cuh"

namespace GLM_CUDA
{
    __host__ __device__ mat3::mat3()
    {
        value = new vec3[3];
        for (int i = 0; i < 3; i++)
        {
            value[i] = vec3();
        }
    }
    __host__ __device__ mat3::mat3(float v0)
    {
        value = new vec3[3];
        for (int i = 0; i < 3; i++)
        {
            value[i] = vec3(v0);
        }
    }
    __host__ __device__ mat3::mat3(vec3 v0, vec3 v1, vec3 v2)
    {
        value = new vec3[3];
        value[0] = v0;
        value[1] = v1;
        value[2] = v2;
    }
    __host__ __device__ mat3::~mat3()
    {
        delete[] value;
    }

    __host__ __device__ vec3& mat3::operator[](unsigned int index)
    {
        return value[index];
    }
    __host__ __device__ const vec3& mat3::operator[] (unsigned int index) const
    {
        return value[index];
    }
    __host__ __device__ mat3& mat3::operator=(const mat3& matrix)
    {
        if (this != &matrix)
        {
            // Prevent memory leak
            delete[] value;
            value = new vec3[3];

            for (int i = 0; i < 3; i++)
            {
                value[i] = matrix[i];
            }

            // Copy data - Using CUDA's memcpy if possible (More efficient, still contain a bug)
            /*
            #ifdef __CUDA_ARCH__
                hipMemcpy(
                    value,
                    matrix,
                    3 * sizeof(vec3),
                    hipMemcpyDeviceToDevice
                );
            #else
                std::copy(matrix[0], matrix[2], value);
            #endif
            */
        }
        return *this;
    }

    __host__ __device__ mat3 mat3::operator+(mat3 matrix)
    {
        mat3 out;
        for (int i = 0; i < 3; ++i)
        {
            out[i] = value[i] + matrix[i];
        }
        return out;
    }
    __host__ __device__ mat3& mat3::operator+=(mat3& matrix)
    {
        for (int i = 0; i < 3; ++i)
        {
            value[i] += matrix[i];
        }
        return *this;
    }

    __host__ __device__ mat3 mat3::operator-(mat3 matrix)
    {
        mat3 out;
        for (int i = 0; i < 3; ++i)
        {
            out[i] = value[i] - matrix[i];
        }
        return out;
    }
    __host__ __device__ mat3& mat3::operator-=(mat3& matrix)
    {
        for (int i = 0; i < 3; ++i)
        {
            value[i] -= matrix[i];
        }
        return *this;
    }

    __host__ __device__ mat3 mat3::operator*(float scalar)
    {
        mat3 out;
        for (int i = 0; i < 3; ++i)
        {
            out[i] = value[i] * scalar;
        }
        return out;
    }
    __host__ __device__ mat3 mat3::operator*(mat3 matrix)
    {
        mat3 out;
        mat3 transposed = transpose(matrix);
        for (int column = 0; column < 3; column++)
        {
            for (int row = 0; row < 3; row++)
            {
                out[column][row] = dot(value[column], transposed[row]);
            }
        }
        return out;
    }
    __host__ __device__ mat3& mat3::operator*=(float scalar)
    {
        for (int i = 0; i < 3; ++i)
        {
            value[i] *= scalar;
        }
        return *this;
    }
    __host__ __device__ mat3& mat3::operator*=(mat3 matrix)
    {
        mat3 transposed = transpose(matrix);
        for (int column = 0; column < 3; column++)
        {
            for (int row = 0; row < 3; row++)
            {
                value[column][row] = dot(value[column], transposed[row]);
            }
        }
        return *this;
    }

    __host__ __device__ mat3 mat3::operator/(float scalar)
    {
        mat3 out;
        for (int i = 0; i < 3; ++i)
        {
            out[i] = value[i] / scalar;
        }
        return out;
    }
    __host__ __device__ mat3& mat3::operator/=(float scalar)
    {
        for (int i = 0; i < 3; ++i)
        {
            value[i] /= scalar;
        }
        return *this;
    }

    __host__ __device__ std::ostream& operator << (std::ostream& os, const mat3& matrix)
    {
        // Expected output
        // ┌ ┌   ┐ ┌   ┐ ┌   ┐ ┐
        // | | a | | d | | g | |
        // | | b | | e | | h | |
        // | | c | | f | | i | |
        // └ └   ┘ └   ┘ └   ┘ ┘

        // Check for maximum length of every number inside matrix
        unsigned int maxLength = 0;
        for (int i = 0; i < 3; i++)
        {
            for (int j = 0; j < 3; j++)
            {
                if (std::to_string(matrix[i][j]).length() > maxLength)
                {
                    maxLength = std::to_string(matrix[i][j]).length();
                }
            }
        }

        for (int row = 0; row < 5; row++)
        {
            std::string leftBracket = (row == 0) ? "┌" : (row == 5) ? "└" : "|";
            std::string rightBracket = (row == 0) ? "┐" : (row == 5) ? "┘" : "|";
            os << leftBracket << " "; 

            for (int column = 0; column < 3; column++)
            {
                std::string number = (
                    (row == 0 || row == 5) ?
                    std::string(maxLength, ' ') :
                    std::string(maxLength - std::to_string(matrix[column][row]).length(), ' ') + std::to_string(matrix[column][row])
                );
                os << leftBracket << " " << number << " " << rightBracket << " ";
            }

            os << rightBracket;
        }
        return os;
    }
}