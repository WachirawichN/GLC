#include "hip/hip_runtime.h"
#include "../../include/vector.cuh"

namespace CUDA_GL
{
    __host__ __device__ vec2 cross(const vec2& a, const vec2& b)
    {
        // [Ax]   [Bx]   [Ay⋅Bx - Ay⋅Bx]
        // [Ay] x [By] = [Ax⋅By − Ax⋅By]
        // Return 0 for vec2
        return vec2();
    }
    __host__ __device__ vec3 cross(const vec3& a, const vec3& b)
    {
        // [Ax]   [Bx]   [Ay⋅Bz - Az⋅By]
        // [Ay] x [By] = [Az⋅Bx − Ax⋅Bz]
        // [Az]   [Bz]   [Ax⋅By − Ay⋅Bx]
        return vec3(
            a[1] * b[2] - a[2] * b[1],
            a[2] * b[0] - a[0] * b[2],
            a[0] * b[1] - a[1] * b[0]
        );
    }
    __host__ __device__ vec4 cross(const vec4& a, const vec4& b)
    {
        // [Ax]   [Bx]   [Ay⋅Bz - Aω⋅Bz]
        // [Ay] x [By] = [Az⋅Bω − Ax⋅Bω]
        // [Az]   [Bz]   [Aω⋅Bx − Ay⋅Bx]
        // [Aω]   [Bω]   [Ax⋅By - Az⋅By]
        return vec4(
            b[2] * (a[1] - a[3]),
            b[3] * (a[2] - a[0]),
            b[0] * (a[3] - a[1]),
            b[1] * (a[0] - a[2])
        );
    }

    __host__ __device__ float dot(const vec2& a, const vec2& b)
    {
        float sum = 0;
        for (int axis = 0; axis < 2; axis++)
        {
            sum += a[axis] * b[axis];
        }
        return sum;
    }
    __host__ __device__ float dot(const vec3& a, const vec3& b)
    {
        float sum = 0;
        for (int axis = 0; axis < 3; axis++)
        {
            sum += a[axis] * b[axis];
        }
        return sum;
    }
    __host__ __device__ float dot(const vec4& a, const vec4& b)
    {
        float sum = 0;
        for (int axis = 0; axis < 4; axis++)
        {
            sum += a[axis] * b[axis];
        }
        return sum;
    }

    __host__ __device__ float length(const vec2& vector)
    {
        #ifdef __CUDA_ARCH__
            return hypotf(vector[0], vector[1]);
        #else
            return std::sqrtf(std::powf(vector[0], 2.0f) + std::powf(vector[1], 2.0f));
        #endif
    }
    __host__ __device__ float length(const vec3& vector)
    {
        #ifdef __CUDA_ARCH__
            return norm3df(vector[0], vector[1], vector[2]);
        #else
            return std::sqrtf(std::powf(vector[0], 2.0f) + std::powf(vector[1], 2.0f) + std::powf(vector[2], 2.0f));
        #endif
    }
    __host__ __device__ float length(const vec4& vector)
    {
        #ifdef __CUDA_ARCH__
            return norm4df(vector[0], vector[1], vector[2], vector[3]);
        #else
            return std::sqrtf(std::powf(vector[0], 2.0f) + std::powf(vector[1], 2.0f) + std::powf(vector[2], 2.0f) + std::powf(vector[3], 2.0f));
        #endif
    }

    __host__ __device__ vec2 normalize(vec2& vector)
    {
        return vector / length(vector);
    }
    __host__ __device__ vec3 normalize(vec3& vector)
    {
        return vector / length(vector);
    }
    __host__ __device__ vec4 normalize(vec4& vector)
    {
        return vector / length(vector);
    }
}