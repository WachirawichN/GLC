#include "../../include/vector.cuh"

namespace GLM_CUDA
{
    __host__ __device__ vec4::vec4()
    {
        value = new float[4];
        for (int i = 0; i < 4; i++)
        {
            value[i] = 0.0f;
        }
    }
    __host__ __device__ vec4::vec4(float v0)
    {
        value = new float[4];
        for (int i = 0; i < 4; i++)
        {
            value[i] = v0;
        }
    }
    __host__ __device__ vec4::vec4(float v0, float v1, float v2, float v3)
    {
        value = new float[4] {v0, v1, v2, v3};
    }
    __host__ __device__ vec4::vec4(const vec4& vector)
    {
        value = new float[4];
        for (int i = 0; i < 4; i++)
        {
            value[i] = vector[i];
        }
    }
    __host__ __device__ vec4::~vec4()
    {
        delete[] value;
    }


    __host__ __device__ float vec4::operator[](unsigned int index) const
    {
        return value[index];
    }
    __host__ __device__ indexProxy vec4::operator[](unsigned int index)
    {
        return indexProxy(value[index]);
    }
    __host__ __device__ vec4& vec4::operator=(const vec4& vector)
    {
        if (this != &vector)
        {
            // Prevent memory leak
            if (value)
            {
                delete[] value;
                value = new float[4];
            }

            for (int i = 0; i < 4; i++)
            {
                value[i] = vector[i];
            }

            // Copy data - Using CUDA's memcpy if possible (More efficient, still contain a bug)
            /*
            #ifdef __CUDA_ARCH__
                hipMemcpy(
                    value,
                    matrix,
                    3 * sizeof(vec3),
                    hipMemcpyDeviceToDevice
                );
            #else
                std::copy(matrix[0], matrix[2], value);
            #endif
            */
        }
        return *this;
    }


    __host__ __device__ vec4 vec4::operator+(vec4 vector)
    {
        vec4 out;
        for (int i = 0; i < 4; i++)
        {
            out[i] = value[i] + vector[i];
        }
        return out;
    }
    __host__ __device__ vec4& vec4::operator+=(vec4& vector)
    {
        for (int i = 0; i < 4; i++)
        {
            value[i] += vector[i];
        }
        return *this;
    }

    __host__ __device__ vec4 vec4::operator-(vec4 vector)
    {
        vec4 out;
        for (int i = 0; i < 4; i++)
        {
            out[i] = value[i] - vector[i];
        }
        return out;
    }
    __host__ __device__ vec4& vec4::operator-=(vec4& vector)
    {
        for (int i = 0; i < 4; i++)
        {
            value[i] -= vector[i];
        }
        return *this;
    }

    __host__ __device__ vec4 vec4::operator*(float scalar)
    {
        vec4 out;
        for (int i = 0; i < 4; i++)
        {
            out[i] = value[i] * scalar;
        }
        return out;
    }
    __host__ __device__ vec4& vec4::operator*=(float scalar)
    {
        for (int i = 0; i < 4; i++)
        {
            value[i] *= scalar;
        }
        return *this;
    }

    __host__ __device__ vec4 vec4::operator/(float scalar)
    {
        vec4 out;
        for (int i = 0; i < 4; i++)
        {
            out[i] = value[i] / scalar;
        }
        return out;
    }
    __host__ __device__ vec4& vec4::operator/=(float scalar)
    {
        for (int i = 0; i < 4; i++)
        {
            value[i] /= scalar;
        }
        return *this;
    }

    __host__ __device__ std::ostream& operator << (std::ostream& os, const vec4& vector)
    {
        return os << "[" << vector[0] << ", " << vector[1] << ", " << vector[2] << ", " << vector[3] << "]";
    }
}